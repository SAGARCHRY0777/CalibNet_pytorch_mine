
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main() {
    int a = 2, b = 7, c;
    int *d_a, *d_b, *d_c;

    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));
    hipMalloc(&d_c, sizeof(int));

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    add<<<1, 1>>>(d_a, d_b, d_c);

    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("Result: %d\n", c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}